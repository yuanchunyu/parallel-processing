

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void 
reduce(float *g, float *o, const int dimx, const int dimy) {

	//extern __shared__ float sdata[];

	unsigned int tid_x = threadIdx.x;
	unsigned int tid_y = threadIdx.y;

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; 

	if (i >= dimx || j >= dimy)
	    return;

	o[i*dimy + j] = g[i*dimy + j] + 1;

	/*sdata[tid_x*blockDim.y + tid_y] = g[i*dimy + j];

	__syncthreads();

	for(unsigned int s_y = blockDim.y/2; s_y > 0; s_y >>= 1)
	{
	    if (tid_y < s_y)
	    {
	        sdata[tid_x * dimy + tid_y] += sdata[tid_x * dimy + tid_y + s_y];
	    }
	    __syncthreads();
	}

	for(unsigned int s_x = blockDim.x/2; s_x > 0; s_x >>= 1 )
	{

	    if(tid_x < s_x)
	    {
	        sdata[tid_x * dimy] += sdata[(tid_x + s_x) * dimy];
	    }
	    __syncthreads();
	}

	float sum;

	if( tid_x == 0 && tid_y == 0)
	{ 
	    sum = sdata[0];
	    atomicAdd (o, sum);   // The result should be the sum of all pixel values. But the program produce 0
	}

//if(tid_x==0 && tid__y == 0 ) 
    //o[blockIdx.x] = sdata[0];

    */
}



int
main()
{   
	int dimx = 320;
	int dimy = 160;
	int num_bytes = dimx*dimy*sizeof(float);

	float *d_a, *h_a, // device and host pointers
	            *d_o=0, *h_o=0;

	h_a = (float*)malloc(num_bytes);
	h_o = (float*)malloc(sizeof(float));

	for (int i=0; i < dimx; i++){   
	    for (int j=0; j < dimy; j++){
	        h_a[i*dimy + j] = 1;
	    }
	}

	hipMalloc( (void**)&d_a, num_bytes );
	hipMalloc( (void**)&d_o, sizeof(int) );

	hipMemcpy( d_a, h_a, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy( d_o, h_o, sizeof(int), hipMemcpyHostToDevice); 

	dim3 grid, block;
	block.x = 4;
	block.y = 4;
	grid.x = dimx / block.x;
	grid.y = dimy / block.y;

	reduce<<<grid, block>>> (d_a, d_o, block.x, block.y);

	std::cout << block.x << " " << block.y << std::endl;
	std::cout << grid.x << " " << grid.y << std::endl;
	std::cout << dimx <<  " " << dimy << " " << dimx*dimy << std::endl;

	hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
	hipMemcpy( h_o, d_o, sizeof(int), hipMemcpyDeviceToHost );

	hipFree(d_a);
	hipFree(d_o);

	for(int i = 0 ; i < dimx ; i++){
		for(int j = 0 ; j < dimy ; j++){
		  cout << "h_a[" << (i*dimy) + j << "]=" << h_a[(i*dimy) + j] << endl;
		}
	}

	for(int i = 0 ; i < dimx ; i++){
		for(int j = 0 ; j < dimy ; j++){
		  cout << "h_o[" << (i*dimy) + j << "]=" << h_o[(i*dimy) + j] << endl;
		}
	}

	
	free(h_a);
	free(h_o);
}
